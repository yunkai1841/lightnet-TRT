
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) {
    return 1.0f / (1.0f + __expf(-x));
}

__global__ void gpuYoloLayerV3(const float* input, float* output,
                               const uint32_t grid_h_, const uint32_t grid_w_,
                               const uint32_t numOutputClasses,
                               const uint32_t numBBoxes) {
    uint32_t x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= grid_w_) || (y_id >= grid_h_) || (z_id >= numBBoxes)) {
        return;
    }

    const int numGridCells = grid_h_ * grid_w_;
    const int bbindex = y_id * grid_w_ + x_id;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)] =
        sigmoidGPU(input[bbindex +
                         numGridCells * (z_id * (5 + numOutputClasses) + 0)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)] =
        sigmoidGPU(input[bbindex +
                         numGridCells * (z_id * (5 + numOutputClasses) + 1)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] =
        __expf(input[bbindex +
                     numGridCells * (z_id * (5 + numOutputClasses) + 2)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] =
        __expf(input[bbindex +
                     numGridCells * (z_id * (5 + numOutputClasses) + 3)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)] =
        sigmoidGPU(input[bbindex +
                         numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    for (uint32_t i = 0; i < numOutputClasses; ++i) {
        output[bbindex +
               numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] =
            sigmoidGPU(
                input[bbindex + numGridCells *
                                    (z_id * (5 + numOutputClasses) + (5 + i))]);
    }
}

// Scaled YOLOv4 format
// https://alexeyab84.medium.com/scaled-yolo-v4-is-the-best-neural-network-for-object-detection-on-ms-coco-dataset-39dfa22fa982
__global__ void gpuYoloV4Layer(const float* input, float* output,
                               const uint32_t grid_h_, const uint32_t grid_w_,
                               const uint32_t numOutputClasses,
                               const uint32_t numBBoxes, float scale_x_y) {
    uint32_t x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= grid_w_) || (y_id >= grid_h_) || (z_id >= numBBoxes)) {
        return;
    }

    const int numGridCells = grid_h_ * grid_w_;
    const int bbindex = y_id * grid_w_ + x_id;
    // bx
    scale_x_y = 1.0;
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)] =
        scale_x_y * (input[bbindex + numGridCells *
                                         (z_id * (5 + numOutputClasses) + 0)]) -
        0.5 * (scale_x_y - 1);
    // by
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)] =
        scale_x_y * (input[bbindex + numGridCells *
                                         (z_id * (5 + numOutputClasses) + 1)]) -
        0.5 * (scale_x_y - 1);
    // bw
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] =
        (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) *
        (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) *
        4;
    // bh
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] =
        (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) *
        (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) *
        4;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)] =
        (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    for (uint32_t i = 0; i < numOutputClasses; ++i) {
        output[bbindex +
               numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] =
            (input[bbindex +
                   numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);
    }
}

hipError_t cudaYoloLayerV3(const void* input, void* output,
                            const uint32_t& batchSize,
                            const uint32_t& n_grid_h_,
                            const uint32_t& n_grid_w_,
                            const uint32_t& numOutputClasses,
                            const uint32_t& numBBoxes, uint64_t outputSize,
                            const uint32_t& new_coords, hipStream_t stream) {
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((n_grid_w_ / threads_per_block.x) + 1,
                          (n_grid_h_ / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    //    printf("YOLOv3 (V4) \n");
    for (int batch = 0; batch < batchSize; ++batch) {
        if (new_coords) {
            gpuYoloV4Layer<<<number_of_blocks, threads_per_block, 0, stream>>>(
                reinterpret_cast<const float*>(input) + (batch * outputSize),
                reinterpret_cast<float*>(output) + (batch * outputSize),
                n_grid_h_, n_grid_w_, numOutputClasses, numBBoxes, 2.0);
        } else {
            gpuYoloLayerV3<<<number_of_blocks, threads_per_block, 0, stream>>>(
                reinterpret_cast<const float*>(input) + (batch * outputSize),
                reinterpret_cast<float*>(output) + (batch * outputSize),
                n_grid_h_, n_grid_w_, numOutputClasses, numBBoxes);
        }
    }
    return hipGetLastError();
}

hipError_t cudaYoloV4Layer(const void* input, void* output,
                            const uint32_t& batchSize,
                            const uint32_t& n_grid_h_,
                            const uint32_t& n_grid_w_,
                            const uint32_t& numOutputClasses,
                            const uint32_t& numBBoxes, uint64_t outputSize,
                            const uint32_t& new_coords, const float scale_x_y,
                            hipStream_t stream) {
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((n_grid_w_ / threads_per_block.x) + 1,
                          (n_grid_h_ / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    //    printf("Scaled YOLOv4 \n");
    for (int batch = 0; batch < batchSize; ++batch) {
        gpuYoloV4Layer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize), n_grid_h_,
            n_grid_w_, numOutputClasses, numBBoxes, scale_x_y);
    }
    return hipGetLastError();
}
